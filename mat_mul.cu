#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

#define TILE_WIDTH 16 

extern "C" void gpu_mat_mul(float* h_M, float* h_N, float* h_P, int M_row, int width, int N_col);

__global__
void gpu_mat_mul_kernel(float* M, float* N, float* P, int M_row, int width, int N_col){

  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x; 
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Identify the row and column of the P element to work on
  // Each thread works on an element of P
  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;

 // if(Row<M_row && Col<N_col){
  int phase_num = ceil(width / (float)TILE_WIDTH);
  float Pvalue = 0;
  // Each thread loads 'Row'th row of M and 'Col'th column of N
  for (int ph = 0; ph < phase_num; ++ph) {

    if((ph*TILE_WIDTH+tx) < width && Row<M_row )     
      Mds[ty][tx] = M[Row * width + ph * TILE_WIDTH + tx];   
    else
      Mds[ty][tx] = 0;
    if((ph*TILE_WIDTH+ty) <width && Col<N_col)
   	 Nds[ty][tx] = N[(ph * TILE_WIDTH + ty) * N_col + Col];
    else 
	Nds[ty][tx] = 0;

    __syncthreads();

    for (int k = 0; k < TILE_WIDTH; ++k) { 
      Pvalue += Mds[ty][k] * Nds[k][tx];
    }

    __syncthreads();
  }
  if(Row<M_row && Col<N_col)
  P[Row * N_col + Col] = Pvalue;
  //}
}

void gpu_mat_mul(float* h_M, float* h_N, float* h_P, int M_row, int width, int N_col) {
  float *d_M, *d_N, *d_P;

  size_t size_of_float = sizeof(float);
  size_t size_M = M_row * width * size_of_float;
  size_t size_N = width * N_col * size_of_float;
  size_t size_P = M_row * N_col * size_of_float;

  hipMalloc((void**)&d_M, size_M);
  hipMalloc((void**)&d_N, size_N);
  hipMalloc((void**)&d_P, size_P);
    
  hipMemcpy(d_M, h_M, size_M, hipMemcpyHostToDevice);
  hipMemcpy(d_N, h_N, size_N, hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  float elapsed_time = 0.0;
    
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  dim3 grid_dim(ceil(N_col/ (float)(TILE_WIDTH)), ceil(M_row/ (float)(TILE_WIDTH)), 1);
  dim3 block_dim(TILE_WIDTH, TILE_WIDTH, 1);
  gpu_mat_mul_kernel<<<grid_dim, block_dim>>>(d_M, d_N, d_P, M_row, width, N_col);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(h_P, d_P, size_P, hipMemcpyDeviceToHost);
    
  // Free device memory for M, N, P
  hipFree(d_M);
  hipFree(d_N);
  hipFree(d_P);

  hipEventElapsedTime(&elapsed_time, start, stop);
    
  printf("  grid  dim:  %d, %d, %d.\n", grid_dim.x, grid_dim.y, grid_dim.z);
  printf("  block dim: %d, %d, %d.\n", block_dim.x, block_dim.y, block_dim.z);
  printf("  kernel time: %.5f sec\n", elapsed_time / 1000);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}

